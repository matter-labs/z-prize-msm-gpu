#include "allocator.cuh"
#include "common.cuh"
#include "msm_kernels.cuh"

namespace allocator {

template <class T> allocation<T>::allocation() : ptr(nullptr), stream(nullptr){};

template <class T> hipError_t allocation<T>::free(hipStream_t new_stream) {
  HANDLE_CUDA_ERROR(hipFreeAsync(ptr, new_stream));
  ptr = nullptr;
  stream = nullptr;
  return hipSuccess;
}

template <class T> hipError_t allocation<T>::free() { return free(stream); }

template <class T> allocation<T>::~allocation() {
  if (ptr != nullptr)
    hipFreeAsync(ptr, stream);
  ptr = nullptr;
  stream = nullptr;
}

template <class T> allocation<T>::operator T *() { return ptr; }

template struct allocation<ff_storage<8u>>;
template struct allocation<ff_storage<12u>>;
template struct allocation<unsigned>;
template struct allocation<void>;
template struct allocation<msm::point_affine>;
template struct allocation<msm::point_projective>;
template struct allocation<msm::point_jacobian>;
template struct allocation<msm::point_xyzz>;

template <class T> size_t get_size_of() { return sizeof(T); }
template <> size_t get_size_of<void>() { return 1; }

template <class T> hipError_t allocate(allocation<T> &allocation, const size_t size, hipMemPool_t pool, hipStream_t stream) {
  T *ptr;
  HANDLE_CUDA_ERROR(hipMallocFromPoolAsync(&ptr, get_size_of<T>() * size, pool, stream));
  allocation.ptr = ptr;
  allocation.stream = stream;
  return hipSuccess;
}

template <class T> hipError_t free(allocation<T> &allocation, hipStream_t stream) { return allocation.free(stream); }

template hipError_t allocate<ff_storage<8u>>(allocation<ff_storage<8u>> &allocation, const size_t size, hipMemPool_t pool, hipStream_t stream);
template hipError_t free<ff_storage<8u>>(allocation<ff_storage<8u>> &allocation, hipStream_t stream);

template hipError_t allocate<ff_storage<12u>>(allocation<ff_storage<12u>> &allocation, const size_t size, hipMemPool_t pool, hipStream_t stream);
template hipError_t free<ff_storage<12u>>(allocation<ff_storage<12u>> &allocation, hipStream_t stream);

template hipError_t allocate<unsigned>(allocation<unsigned> &allocation, const size_t size, hipMemPool_t pool, hipStream_t stream);
template hipError_t free<unsigned>(allocation<unsigned> &allocation, hipStream_t stream);

template hipError_t allocate<void>(allocation<void> &allocation, const size_t size, hipMemPool_t pool, hipStream_t stream);
template hipError_t free<void>(allocation<void> &allocation, hipStream_t stream);

template hipError_t allocate<msm::point_affine>(allocation<msm::point_affine> &allocation, const size_t size, hipMemPool_t pool, hipStream_t stream);
template hipError_t free<msm::point_affine>(allocation<msm::point_affine> &allocation, hipStream_t stream);

template hipError_t allocate<msm::point_projective>(allocation<msm::point_projective> &allocation, const size_t size, hipMemPool_t pool, hipStream_t stream);
template hipError_t free<msm::point_projective>(allocation<msm::point_projective> &allocation, hipStream_t stream);

template hipError_t allocate<msm::point_jacobian>(allocation<msm::point_jacobian> &allocation, const size_t size, hipMemPool_t pool, hipStream_t stream);
template hipError_t free<msm::point_jacobian>(allocation<msm::point_jacobian> &allocation, hipStream_t stream);

template hipError_t allocate<msm::point_xyzz>(allocation<msm::point_xyzz> &allocation, const size_t size, hipMemPool_t pool, hipStream_t stream);
template hipError_t free<msm::point_xyzz>(allocation<msm::point_xyzz> &allocation, hipStream_t stream);

} // namespace allocator
